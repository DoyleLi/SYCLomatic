// RUN: dpct -out-root %T/driver-stream-and-event %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-stream-and-event/driver-stream-and-event.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/driver-stream-and-event/driver-stream-and-event.dp.cpp -o %T/driver-stream-and-event/driver-stream-and-event.dp.o %}
#include "hip/hip_runtime.h"
#include <vector>
// CHECK: #include <future>
template <typename T>
// CHECK: void my_error_checker(T ReturnValue, char const *const FuncName) {
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

void foo(){
  hipFunction_t f;
  hipStream_t s;
  hipEvent_t e;

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuFuncSetCacheConfig was removed because SYCL currently does not support configuring shared memory on devices.
  //CHECK-NEXT: */
  cuFuncSetCacheConfig(f, hipFuncCachePreferNone);

  //CHECK: s = dpct::get_current_device().create_queue();
  hipStreamCreateWithFlags(&s, hipStreamDefault);

  // CHECK: int streamStatus = DPCT_CHECK_ERROR((s->ext_oneapi_empty())); 
  // CHECK-NEXT: if (streamStatus == 0);
  hipError_t streamStatus = hipStreamQuery(s);
  if (streamStatus == hipSuccess);

  //CHECK: s->wait();
  hipStreamSynchronize(s);

  // CHECK: if (DPCT_CHECK_ERROR((s->ext_oneapi_empty())) == 0);
  if (hipStreamQuery(s) == hipSuccess);

  //CHECK: s->ext_oneapi_submit_barrier({*e});
  hipEventCreateWithFlags(&e, hipEventDefault);
  hipStreamWaitEvent(s, e, 0);

  //CHECK: /*
  //CHECK-NEXT: DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  //CHECK-NEXT: */
  //CHECK-NEXT: e_ct1 = std::chrono::steady_clock::now();
  //CHECK-NEXT: *e = s->ext_oneapi_submit_barrier();
  //CHECK-NEXT: e->wait_and_throw();
  hipEventRecord(e, s);
  hipEventSynchronize(e);

  //CHECK: sycl::info::event_command_status r;
  //CHECK-NEXT: r = e->get_info<sycl::info::event::command_execution_status>();
  hipError_t r;
  r = hipEventQuery(e);

  //CHECK: dpct::event_ptr start, end;
  //CHECK-NEXT: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
  //CHECK-NEXT: std::chrono::time_point<std::chrono::steady_clock> end_ct1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  //CHECK-NEXT: */
  //CHECK-NEXT: start_ct1 = std::chrono::steady_clock::now();
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  //CHECK-NEXT: */
  //CHECK-NEXT: end_ct1 = std::chrono::steady_clock::now();
  //CHECK-NEXT: float result_time;
  //CHECK-NEXT: result_time = std::chrono::duration<float, std::milli>(end_ct1 - start_ct1).count();
  hipEvent_t start, end;
  hipEventRecord(start, s);
  hipEventRecord(end, s);
  hipEventSynchronize(start);
  hipEventSynchronize(end);
  float result_time;
  hipEventElapsedTime(&result_time, start, end);

  int rr;
  //CHECK: rr = dpct::get_kernel_function_info(f).max_work_group_size;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, f);

  //CHECK: /*
  //CHECK-NEXT: DPCT1132:{{[0-9]+}}: SYCL 2020 does not support accessing the statically allocated shared memory for the kernel. The API is replaced with member variable "shared_size_bytes". Please set the appropriate value for "shared_size_bytes".
  //CHECK-NEXT: */
  //CHECK: rr = dpct::get_kernel_function_info(f).shared_size_bytes /* statically allocated shared memory per work-group in bytes */;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, f);

  //CHECK: /*
  //CHECK-NEXT: DPCT1132:{{[0-9]+}}: SYCL 2020 does not support accessing the local memory for the kernel. The API is replaced with member variable "local_size_bytes". Please set the appropriate value for "local_size_bytes".
  //CHECK-NEXT: */
  //CHECK: rr = dpct::get_kernel_function_info(f).local_size_bytes /* local memory per work-item in bytes */;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_LOCAL_SIZE_BYTES, f);

  //CHECK: /*
  //CHECK-NEXT: DPCT1132:{{[0-9]+}}: SYCL 2020 does not support accessing the memory size of user-defined constants for the kernel. The API is replaced with member variable "const_size_bytes". Please set the appropriate value for "const_size_bytes".
  //CHECK-NEXT: */
  //CHECK: rr = dpct::get_kernel_function_info(f).const_size_bytes /* user-defined constant kernel memory in bytes */;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_CONST_SIZE_BYTES, f);

  //CHECK: /*
  //CHECK-NEXT: DPCT1132:{{[0-9]+}}: SYCL 2020 does not support accessing the required number of registers for the kernel. The API is replaced with member variable "num_regs". Please set the appropriate value for "num_regs".
  //CHECK-NEXT: */
  //CHECK: rr = dpct::get_kernel_function_info(f).num_regs /* number of registers for each thread */;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_NUM_REGS, f);

  hipEventDestroy(start);

  hipEventDestroy(end);
}


// CHECK: void process(dpct::queue_ptr st, char *data, int status) {}
void process(hipStream_t st, char *data, hipError_t status) {}

template<typename T>
// CHECK: void callback(dpct::queue_ptr hStream, int status, void *userData) {
void callback(hipStream_t hStream, hipError_t status, void* userData) {
  T *data = static_cast<T *>(userData);
  process(hStream, data, status);
}

void test_stream() {
  hipStream_t hStream;
  void* data;
  unsigned int flag;
  size_t length;
  hipDeviceptr_t  cuPtr;
  // CHECK: std::async([&]() { hStream->wait(); callback<char>(hStream, 0, data); });
  hipStreamAddCallback(hStream, callback<char>, data, flag);

  // CHECK: int result = DPCT_CHECK_ERROR(std::async([&]() { hStream->wait(); callback<char>(hStream, 0, data); }));
  hipError_t result = hipStreamAddCallback(hStream, callback<char>, data, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuStreamAttachMemAsync was removed because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  hipStreamAttachMemAsync(hStream, cuPtr, length, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cuStreamAttachMemAsync was replaced with 0 because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  //CHECK-NEXT: result = 0;
  result = hipStreamAttachMemAsync(hStream, cuPtr, length, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cuStreamAttachMemAsync was replaced with 0 because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  //CHECK-NEXT: MY_ERROR_CHECKER(0);
  MY_ERROR_CHECKER(hipStreamAttachMemAsync(hStream, cuPtr, length, flag));

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuStreamAttachMemAsync was removed because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  hipStreamAttachMemAsync(hStream, cuPtr, std::vector<int>(1,1).front(), flag);

  // CHECK: dpct::get_current_device().destroy_queue(hStream);
  hipStreamDestroy(hStream);
}

void test_cuEventRecord_crash(hipEvent_t hEvent, hipStream_t hStream)
{
  // CHECK: int result = DPCT_CHECK_ERROR(*(dpct::event_ptr)hEvent = ((dpct::queue_ptr)hStream)->ext_oneapi_submit_barrier());
  hipError_t result = hipEventRecord((hipEvent_t)hEvent, (hipStream_t)hStream);
}

unsigned getEventFlags(bool enabledSyncBlock) {
  // CHECK: /*
  // CHECK-NEXT: DPCT1014:{{[0-9]+}}: The CU_EVENT_DISABLE_TIMING is not supported for SYCL event. The output parameter(s) is set to 0.
  // CHECK-NEXT: */
  // CHECK-NEXT: unsigned flags = 0;
  unsigned flags = hipEventDisableTiming;

  if (enabledSyncBlock)
    // CHECK: /*
    // CHECK-NEXT: DPCT1014:{{[0-9]+}}: The CU_EVENT_BLOCKING_SYNC is not supported for SYCL event. The output parameter(s) is set to 0.
    // CHECK-NEXT: */
    // CHECK-NEXT: flags |= 0;
    flags |= hipEventBlockingSync;

  return flags;
}
